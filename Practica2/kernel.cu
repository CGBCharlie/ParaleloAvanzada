#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void idx_calc_tid(int* input) {
    int tid = threadIdx.x;
    printf("[DEVICE] threadIdx.x: %d, data: %d\n\r", tid, input[tid]);
}

__global__ void idx_calc_gid(int* input) {
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x;
    int gid = tid + offset;
    printf("[DEVICE] blockIdx.x: %d, threadIdx.x: %d, gid: %d, data: %d\n\r", blockIdx.x, tid, gid, input[gid]);
}

__global__ void idx_calc_gid2D(int* input) {
    int tid = threadIdx.x;
    int offsetBlock = blockIdx.x * blockDim.x;
    int offsetRow = blockIdx.y * blockDim.x * gridDim.x;
    int gid = tid + offsetBlock + offsetRow;
    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d, data: %d\n\r", gridDim.x, blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

__global__ void idx_calc_gid_2D_2(int* input) {
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int offsetBlock = blockIdx.x * blockDim.x * blockDim.y;
    int offsetRow = blockIdx.y * blockDim.x * blockDim.y * gridDim.x;
    int gid = tid + offsetBlock + offsetRow;
    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, threadIdx.y: %d, gid: %d, data: %d\n\r", gridDim.x, blockIdx.x, blockIdx.y, tid, threadIdx.y, gid, input[gid]);
}

int main()
{
    const int N = 16;

    int a[N] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16 };

    int size = N * sizeof(int);

    int* d_a = 0;

    hipMalloc((void**)&d_a, size);;

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // ## act1 ## //
    //idx_calc_tid << <1, N >> > (d_a);
    
    // ## act2 ## //
    //idx_calc_gid << <2, 8 >> > (d_a);

    // ## act3 ## //
    //idx_calc_gid << <4, 4 >> > (d_a);

    // ## act4 ## //
    /*dim3 grid(2, 2);
    dim3 block(4);
    idx_calc_gid2D << <grid, block >> > (d_a);*/

    // ## act5 ## //    
    dim3 grid(2, 2);
    dim3 block(2, 2);
    idx_calc_gid_2D_2<< <grid, block >> > (d_a);

    hipDeviceSynchronize();

    hipDeviceReset();

    hipFree(d_a);
    return 0;
}
